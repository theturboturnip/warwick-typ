//
// Created by samuel on 18/02/2021.
//

#include "CudaGraphCapture.cuh"

CudaGraphCapture::~CudaGraphCapture() {
    if (graph.has_value() && graph.get()) {
        hipGraphDestroy(graph.release());
    }
    if (instance.has_value() && instance.get()) {
        hipGraphExecDestroy(instance.release());
    }
}

void CudaGraphCapture::record(std::function<void()> record) {
    DASSERT(!recorded);
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

    record();

    hipGraph_t capturedGraph = nullptr;
    hipStreamEndCapture(stream, &capturedGraph);
    DASSERT(capturedGraph != nullptr);

    graph.set(capturedGraph);

    hipGraphExec_t toExec = nullptr;
    hipGraphInstantiate(&toExec, graph.get(), nullptr, nullptr, 0);
    instance.set(toExec);

    recorded = true;
}

void CudaGraphCapture::execute() {
    hipGraphLaunch(instance.get(), stream);
}
