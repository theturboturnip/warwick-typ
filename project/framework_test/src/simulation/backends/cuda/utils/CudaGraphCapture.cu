//
// Created by samuel on 18/02/2021.
//

#include "CudaGraphCapture.cuh"

CudaGraphCapture::~CudaGraphCapture() {
    if (graph.has_value() && graph.get()) {
        hipGraphDestroy(graph.get());
        graph.set(nullptr);
    }
    if (instance.has_value() && instance.get()) {
        hipGraphExecDestroy(instance.get());
        instance.set(nullptr);
    }
}

void CudaGraphCapture::recordOrExecute(std::function<void()> record) {
    if (!recorded) {
        hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

        record();

        hipGraph_t capturedGraph = nullptr;
        hipStreamEndCapture(stream, &capturedGraph);
        DASSERT(capturedGraph != nullptr);

        graph.set(capturedGraph);

        hipGraphExec_t toExec = nullptr;
        hipGraphInstantiate(&toExec, graph.get(), nullptr, nullptr, 0);
        instance.set(toExec);

        recorded = true;
    }
    hipGraphLaunch(instance.get(), stream);
}
