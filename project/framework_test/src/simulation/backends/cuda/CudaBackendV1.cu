#include "hip/hip_runtime.h"
//
// Created by samuel on 12/08/2020.
//

#include "CudaBackendV1.cuh"
#include <simulation/backends/cuda/kernels/redblack.cuh>

#include "simulation/backends/original/simulation.h"
#include "simulation/backends/cuda/kernels/simple.cuh"

inline float host_min(float x, float y) {
    return (x<y) ? x : y;
}

inline float host_max(float x, float y) {
    return (x>y) ? x : y;
}

template<bool UnifiedMemory>
CudaBackendV1<UnifiedMemory>::CudaBackendV1(SimulationAllocs allocs, const FluidParams& params, const SimSnapshot& s)
    : params(params),
      simSize(s.simSize),
      matrix_size(simSize.pixel_size.x + 2, simSize.pixel_size.y + 2),
      redblack_matrix_size(matrix_size.x, matrix_size.y / 2),

      imax(simSize.pixel_size.x),
      jmax(simSize.pixel_size.y),
      x_length(simSize.physical_size.x),
      y_length(simSize.physical_size.y),
      del_x(simSize.del_x()),
      del_y(simSize.del_y()),
      ibound(s.get_boundary_cell_count()),
      ifluid(imax * jmax - ibound),

      u(allocs.alloc, allocs.u),
      v(allocs.alloc, allocs.v),

      f(allocs.alloc, matrix_size),
      g(allocs.alloc, matrix_size),

      p(allocs.alloc, allocs.p),
      p_buffered(allocs.alloc, matrix_size),
      p_sum_squares(allocs.alloc, matrix_size),

      p_beta(allocs.alloc, matrix_size),

      rhs(allocs.alloc, matrix_size),
      flag(allocs.alloc, matrix_size),
      fluidmask(allocs.alloc, allocs.fluidmask),
      surroundmask(allocs.alloc, matrix_size),

      reducer_fullsize(allocs.alloc, u.raw_length)
{
    flag.memcpy_in(s.get_legacy_cell_flags());

    rhs.zero_out();

    f.zero_out();
    g.zero_out();

    hipStreamCreate(&stream);

    // TODO - Remove this
    //static_assert(UnifiedMemory, "CUDA backend currently requires UnifiedMemory for initial work to function");

    // Split pressure to red/black in preparation for poisson, which only operates on split matrices
    OriginalOptimized::splitToRedBlack(p.joined.as_cpu(),
                                       p.red.as_cpu(), p.black.as_cpu(),
                                       imax, jmax);
    p_buffered.red.memcpy_in(p.red);
    p_buffered.black.memcpy_in(p.black);


    // TODO - remove poisson_error_threshold from args
    OriginalOptimized::calculatePBeta(p_beta.joined.as_cpu(), flag.as_cpu(),
                                      imax, jmax, del_x, del_y,
                                      params.poisson_error_threshold, params.poisson_omega);
    OriginalOptimized::splitToRedBlack(p_beta.joined.as_cpu(),
                                       p_beta.red.as_cpu(), p_beta.black.as_cpu(),
                                       imax, jmax);

    // Calculate the fluidmask and surroundedmask items
//    OriginalOptimized::calculateFluidmask((int**)fluidmask.as_cpu(), (const char**)flag.as_cpu(), imax, jmax);
    OriginalOptimized::splitFluidmaskToSurroundedMask((const int **)(fluidmask.as_cpu()),
                                                      (int**)surroundmask.red.as_cpu(), (int**)surroundmask.black.as_cpu(),
                                                      imax, jmax);

    int dstDevice = -1;
    hipGetDevice(&dstDevice);// TODO
    hipDeviceProp_t thisDevice;
    hipGetDeviceProperties(&thisDevice, dstDevice);
    printf("device num: %d device name: %s\n", dstDevice, thisDevice.name);
    u.dispatch_gpu_prefetch(dstDevice, stream);
    v.dispatch_gpu_prefetch(dstDevice, stream);
    p.dispatch_gpu_prefetch(dstDevice, stream);
    p_buffered.dispatch_gpu_prefetch(dstDevice, stream);
    rhs.dispatch_gpu_prefetch(dstDevice, stream);
    f.dispatch_gpu_prefetch(dstDevice, stream);
    g.dispatch_gpu_prefetch(dstDevice, stream);
    p_beta.dispatch_gpu_prefetch(dstDevice, stream);

    hipStreamSynchronize(stream);
}

template<bool UnifiedMemory>
CudaBackendV1<UnifiedMemory>::~CudaBackendV1() {
    hipStreamDestroy(stream);
}


template<bool UnifiedMemory>
float CudaBackendV1<UnifiedMemory>::findMaxTimestep() {
    float delta_t = -1;
    auto fabsf_lambda = [] __device__ (float x) { return fabsf(x); };
    auto max_lambda = [] __device__ (float x, float y) { return max(x, y); };
    // TODO - having multiple reducers here would be more efficient - could dispatch both, and then wait for one then the other?
    float u_max = reducer_fullsize.map_reduce(u, fabsf_lambda, max_lambda, stream);
    u_max = host_max(u_max, 1.0e-10);
    float v_max = reducer_fullsize.map_reduce(v, fabsf_lambda, max_lambda, stream);
    v_max = host_max(v_max, 1.0e-10);

    float delt_u = del_x/u_max;
    float delt_v = del_y/v_max;
    // This used to be deltRe = 1/(1/(delx*delx)+1/(dely*dely))*Re/2.0;
    // the original version has 2.0 at the end, but this only ends up doing the rest of the equation, promoting it to double, dividing it, and demoting back to int.
    // this is equivalent to dividing by 2.0f without any double-promotions.
    float deltRe = 1.0f/(1.0f/(del_x*del_x)+1.0f/(del_y*del_y))*params.Re/2.0f;

    if (delt_u<delt_v) {
        delta_t = host_min(delt_u, deltRe);
    } else {
        delta_t = host_min(delt_v, deltRe);
    }
    delta_t = params.timestep_safety * (delta_t); // multiply by safety factor

//    printf("GPU del_t\n");
//    printf("u_max: %a\tv_max: %a\n", u_max, v_max);
//    printf("delt_u: %a\tdelt_v: %a\tdelt_re: %a\n", delt_u, delt_v, deltRe);
//    printf("delta_t: %a\n", delta_t);

//    float cpu_delta_t = -1;
//    OriginalOptimized::setTimestepInterval(&cpu_delta_t,
//                        imax, jmax,
//                        del_x, del_y,
//                        u.as_cpu(), v.as_cpu(),
//                        params.Re,
//                        params.timestep_safety
//                        );

//    printf("CPU del_t\ndelta_t: %a\n", delta_t);

    DASSERT(delta_t != -1);
    return delta_t;
}

template<bool UnifiedMemory>
void CudaBackendV1<UnifiedMemory>::tick(float timestep) {
    auto gpu_params = CommonParams{
            .size = uint2{matrix_size.x, matrix_size.y},
            .redblack_size = uint2{redblack_matrix_size.x, redblack_matrix_size.y},
            .col_pitch_4byte=u.col_pitch,
            .col_pitch_redblack=rhs.red.col_pitch,
            .deltas = float2{del_x, del_y},
            .timestep = timestep,
    };
    dim3 blocksize_2d(1, 64);
    dim3 gridsize_2d(
            (matrix_size.x + blocksize_2d.x - 1) / blocksize_2d.x,
            (matrix_size.y + blocksize_2d.y - 1) / blocksize_2d.y
            );

    dim3 blocksize_redblack(1, 64);
    dim3 gridsize_redblack(
            (redblack_matrix_size.x + blocksize_redblack.x - 1) / blocksize_redblack.x,
            (redblack_matrix_size.y + blocksize_redblack.y - 1) / blocksize_redblack.y
    );
    //printf("blksize_redblack: %d %d, gridsize: %d %d\n", blocksize_redblack.x, blocksize_redblack.y, gridsize_redblack.x, gridsize_redblack.y);

    dim3 blocksize_vertical(32);
    dim3 gridsize_vertical((matrix_size.y + blocksize_vertical.x - 1) / blocksize_vertical.x);

    dim3 blocksize_horizontal(32);
    dim3 gridsize_horizontal((matrix_size.x + blocksize_horizontal.x - 1) / blocksize_horizontal.x);

    computeTentativeVelocity_apply<<<gridsize_2d, blocksize_2d, 0, stream>>>(
            u.as_gpu(), v.as_gpu(), fluidmask.as_gpu(),
            f.as_gpu(), g.as_gpu(),
            gpu_params, params.gamma, params.Re
            );

    computeTentativeVelocity_postproc_vertical<<<gridsize_vertical, blocksize_vertical, 0, stream>>>(u.as_gpu(), f.as_gpu(), gpu_params);
    computeTentativeVelocity_postproc_horizontal<<<gridsize_horizontal, blocksize_horizontal, 0, stream>>>(v.as_gpu(), g.as_gpu(), gpu_params);

//    OriginalOptimized::computeRhs(f.as_cpu(), g.as_cpu(), rhs2.as_cpu(), flag.as_cpu(),
//               imax, jmax, timestep, del_x, del_y);

    computeRHS_1per<<<gridsize_2d, blocksize_2d, 0, stream>>>(f.as_gpu(), g.as_gpu(), fluidmask.as_gpu(), rhs.joined.as_gpu(), gpu_params);
    //hipStreamSynchronize(stream);


    if (ifluid > 0) {
        constexpr bool UseCPUPoisson = false;
        if constexpr (UnifiedMemory && UseCPUPoisson) {
            OriginalOptimized::poissonSolver<false>(p.joined.as_cpu(), p.red.as_cpu(), p.black.as_cpu(),
                                                    p_beta.joined.as_cpu(), p_beta.red.as_cpu(), p_beta.black.as_cpu(),
                                                    rhs.joined.as_cpu(), rhs.red.as_cpu(), rhs.black.as_cpu(),
                                                    (int **) fluidmask.as_cpu(), (int **) surroundmask.black.as_cpu(),
                                                    flag.as_cpu(), imax, jmax,
                                                    del_x, del_y,
                                                    params.poisson_error_threshold, params.poisson_max_iterations, params.poisson_omega,
                                                    ifluid);
        } else {
            // Sum of squares of pressure - reduction
            // poisson_pSquareSumReduce(p.joined.as_gpu(), p_sum_squares.as_gpu())
            // p0 = p_sum_squares.as_cpu(?????)???
            // TODO - accessing memory like this is very convenient with managed memory
            //  We *might* be able to us VK_EXT_external_memory_host to import CUDA Managed Memory as Vulkan, bypassing Vulkan allocations

            //const float partial_res_sqr_thresh = params.poisson_error_threshold * p0 * params.poisson_error_threshold * p0 * (float)ifluid;

            // Split RHS
            dispatch_splitRedBlackCUDA(rhs, gridsize_2d, blocksize_2d, gpu_params);
            // [NO CUDA STREAM SYNC NECESSARY]
            // hipStreamSynchronize(stream);

            // Red/Black SOR-iteration
            for (int iter = 0; iter < params.poisson_max_iterations; iter++) {
            //  redblack<Red>();
                dispatch_poissonRedBlackCUDA<RedBlack::Red>(gridsize_redblack, blocksize_redblack, iter, gpu_params);
            //  float approxRes = redblack<Black>(); (capture approximate residual here)
                //float approxRes; // TODO - ???
                dispatch_poissonRedBlackCUDA<RedBlack::Black>(gridsize_redblack, blocksize_redblack, iter, gpu_params);//&approxRes);
            //  [ IMPLICIT STREAM SYNC FOR RESIDUAL ]
                // [ NOT NECESSARY WHEN NOT CALCULATING RESIDUAL ]
            //  if (approxRes < partial_res_sqr_thresh)
            //      TODO - necessary to capture full res at all? if the approxRes is actually accurate, then maybe not
            //       If we have to calculate this we may have to merge pressure here
            //      break;
            //  TODO - dynamic error
            }

            // join p
            dispatch_joinRedBlackCUDA(p, gridsize_2d, blocksize_2d, gpu_params);
            // Stream sync not necessary here, because the rest is CUDA
        }
    }

//    OriginalOptimized::updateVelocity(u.as_cpu(), v.as_cpu(),
//                       f.as_cpu(), g.as_cpu(),
//                       p.as_cpu(), flag.as_cpu(),
//                       imax, jmax, timestep, del_x, del_y);
    updateVelocity_1per<<<gridsize_2d, blocksize_2d, 0, stream>>>(f.as_gpu(), g.as_gpu(), p.joined.as_gpu(), fluidmask.as_gpu(),
                                                                      u.as_gpu(), v.as_gpu(),
                                                                      gpu_params);

    boundaryConditions_preproc_vertical<<<gridsize_vertical, blocksize_vertical, 0, stream>>>( u.as_gpu(),  v.as_gpu(), gpu_params);
    boundaryConditions_preproc_horizontal<<<gridsize_horizontal, blocksize_horizontal, 0, stream>>>( u.as_gpu(),  v.as_gpu(), gpu_params);

    boundaryConditions_apply<<<gridsize_2d, blocksize_2d, 0, stream>>>( flag.as_gpu(),
                                                                           u.as_gpu(),  v.as_gpu(),
                                                                           gpu_params);

    boundaryConditions_inputflow_west_vertical<<<gridsize_vertical, blocksize_vertical, 0, stream>>>(
            u.as_gpu(),  v.as_gpu(),
            float2{params.initial_velocity_x, params.initial_velocity_y},
            gpu_params
            );

//    OriginalOptimized::applyBoundaryConditions(u2.as_cpu(), v2.as_cpu(), flag.as_cpu(), imax, jmax, params.initial_velocity_x, params.initial_velocity_y);
}

template<bool UnifiedMemory>
void CudaBackendV1<UnifiedMemory>::dispatch_splitRedBlackCUDA(CudaUnifiedRedBlackArray<float, UnifiedMemory, RedBlackStorage::WithJoined>& to_split,
                                               dim3 gridsize_2d, dim3 blocksize_2d,
                                               CommonParams params)
{
    split_redblack_simple<<<gridsize_2d, blocksize_2d, 0, stream>>>(
            to_split.joined.as_gpu(),
            to_split.red.as_gpu(), to_split.black.as_gpu(),
            params
    );
}
template<bool UnifiedMemory>
void CudaBackendV1<UnifiedMemory>::dispatch_joinRedBlackCUDA(CudaUnifiedRedBlackArray<float, UnifiedMemory, RedBlackStorage::WithJoined>& to_join,
                                              dim3 gridsize_2d, dim3 blocksize_2d,
                                              CommonParams params)
{
    join_redblack_simple<<<gridsize_2d, blocksize_2d,0, stream>>>(
            to_join.red.as_gpu(), to_join.black.as_gpu(),
            to_join.joined.as_gpu(),
            params
    );
}

template<bool UnifiedMemory>
template<RedBlack Kind>
void CudaBackendV1<UnifiedMemory>::dispatch_poissonRedBlackCUDA(dim3 gridsize_redblack, dim3 blocksize_redblack, int iter, CommonParams gpu_params) {
    // TODO - Use HALF SIZE dimensions! the poisson kernel operates on redblack ONLY

    // For a p_red computation: do p_red/p_buffered_black into p_buffered_red, while copying p_buffered_black into p_black.
    // Modern Nvidia GPUs can do parallel memcpy and compute, so this shouldn't take longer

    // Note - all p.get<Kind> functions need to have the template specifier: "p.template get<Kind>()".
    // This is because:
    //  1. CudaBackendV1 is templated on UnifiedMemory
    //  2. Which causes the types of p, p_buffered, rhs etc. to be *dependent* on the value of UnifiedMemory.
    //  3. Because of template specializations, the C++ compiler can't know ahead of time if p.get will be a template function or a data member.
    //  4. This means the C++ compiler can't tell what p.get<Kind>() means - is it (p.get) < Kind > ()? or is it a template function invocation?
    //  5. Using the '.template' syntax tells the C++ compiler to explicitly expect a template function here. It's ugly as hell, but it works.
    // The way to fix this is to stop using shitty languages like C++. However, given the state of CUDA on better languages like Rust is kinda poor,
    // this isn't feasible right now.

    constexpr bool DoubleBuffer = false;

    if (DoubleBuffer) {
        poisson_single_tick<<<gridsize_redblack, blocksize_redblack, 0, stream>>>(
                p.template get<Kind>().as_gpu(),
                p_buffered.template get_other<Kind>().as_gpu(),
                rhs.template get<Kind>().as_gpu(),
                p_beta.template get<Kind>().as_gpu(),

                p_buffered.template get<Kind>().as_gpu(),

                (Kind == RedBlack::Black) ? 1 : 0,// 0 if red, 1 if black

                params.poisson_omega,

                iter,

                gpu_params);

        // TODO - this needs to be done in a separate stream to overlap
        p.template get_other<Kind>().dispatch_memcpy_in(p_buffered.template get_other<Kind>(), stream);
    } else {
        poisson_single_tick<<<gridsize_redblack, blocksize_redblack, 0, stream>>>(
                p.template get<Kind>().as_gpu(),
                p.template get_other<Kind>().as_gpu(),
                rhs.template get<Kind>().as_gpu(),
                p_beta.template get<Kind>().as_gpu(),

                p.template get<Kind>().as_gpu(),

                (Kind == RedBlack::Black) ? 1 : 0,// 0 if red, 1 if black

                params.poisson_omega,

                iter,

                gpu_params);
    }

//    hipError_t error = (hipPeekAtLastError());
//    if (error != hipSuccess) {
//        FATAL_ERROR("CUDA ERROR %s\n", hipGetErrorString(error));
//    }
}

template<>
LegacySimDump CudaBackendV1<true>::dumpStateAsLegacy() {
    hipStreamSynchronize(stream);
    auto dump = LegacySimDump(simSize.to_legacy());
    dump.u = u.extract_data();
    dump.v = v.extract_data();
    dump.p = p.joined.extract_data();
    dump.flag = flag.extract_data();
    return dump;
}
//template<>
//LegacySimDump CudaBackendV1<false>::dumpStateAsLegacy() {
//    static_assert(false, "Cannot dump state, not using Unified Memory");
//}
template<bool UnifiedMemory>
SimSnapshot CudaBackendV1<UnifiedMemory>::get_snapshot() {
    return SimSnapshot::from_legacy(dumpStateAsLegacy());
}


template class CudaBackendV1<true>;
template class CudaBackendV1<false>;